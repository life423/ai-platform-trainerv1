#include <hip/hip_runtime.h>
#include <iostream>

__global__ void dummyKernel(float* data) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx == 0) data[0] = 3.1415f;
}

int main() {
    // 1. Device count
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);
    if (err != hipSuccess || deviceCount == 0) {
        std::cerr << "No CUDA device found: " << hipGetErrorString(err) << "\n";
        return 1;
    }
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    std::cout << "Using GPU: " << prop.name << "\n";

    // 2. Allocate & launch kernel
    float* d_val = nullptr;
    hipMalloc(&d_val, sizeof(float));
    hipMemset(d_val, 0, sizeof(float));

    // Timing events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    dummyKernel<<<1, 32>>>(d_val);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float ms = 0;
    hipEventElapsedTime(&ms, start, stop);

    // 3. Copy back and print
    float h_val = 0;
    hipMemcpy(&h_val, d_val, sizeof(float), hipMemcpyDeviceToHost);
    std::cout << "Kernel time: " << ms << " ms, result: " << h_val << "\n";

    // 4. Memory info
    size_t freeMem = 0, totalMem = 0;
    hipMemGetInfo(&freeMem, &totalMem);
    std::cout << "GPU Memory Used: "
              << (totalMem - freeMem) / 1024 / 1024 << " MB / "
              << totalMem / 1024 / 1024 << " MB\n";

    // Cleanup
    hipFree(d_val);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return 0;
}
