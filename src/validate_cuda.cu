#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>

// Simple CUDA kernel to validate CUDA is working
__global__ void validateCudaKernel(int *result) {
    // This kernel can use threadIdx and blockIdx
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx == 0) {
        *result = 42;  // Magic number to verify kernel ran
    }
}

// Function to check CUDA error
#define CUDA_CHECK(call) \
    do { \
        hipError_t error = call; \
        if (error != hipSuccess) { \
            std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__ \
                     << " code=" << error << " \"" << hipGetErrorString(error) << "\"" << std::endl; \
            exit(1); \
        } \
    } while(0)

int main() {
    std::cout << "=== CUDA Validation Test ===" << std::endl;
    
    // Check if CUDA is available
    int deviceCount = 0;
    hipError_t error = hipGetDeviceCount(&deviceCount);
    
    if (error != hipSuccess) {
        std::cerr << "CUDA Error: " << hipGetErrorString(error) << std::endl;
        return 1;
    }
    
    if (deviceCount == 0) {
        std::cerr << "No CUDA devices found!" << std::endl;
        return 1;
    }
    
    std::cout << "Found " << deviceCount << " CUDA device(s)" << std::endl;
    
    // Get device properties
    for (int i = 0; i < deviceCount; i++) {
        hipDeviceProp_t prop;
        CUDA_CHECK(hipGetDeviceProperties(&prop, i));
        
        std::cout << "\nDevice " << i << ": " << prop.name << std::endl;
        std::cout << "  Compute capability: " << prop.major << "." << prop.minor << std::endl;
        std::cout << "  Total memory: " << prop.totalGlobalMem / (1024*1024) << " MB" << std::endl;
        std::cout << "  Multiprocessors: " << prop.multiProcessorCount << std::endl;
    }
    
    // Test kernel execution
    std::cout << "\nTesting kernel execution..." << std::endl;
    
    int h_result = 0;
    int *d_result;
    
    // Allocate device memory
    CUDA_CHECK(hipMalloc(&d_result, sizeof(int)));
    CUDA_CHECK(hipMemcpy(d_result, &h_result, sizeof(int), hipMemcpyHostToDevice));
    
    // Launch kernel
    validateCudaKernel<<<1, 1>>>(d_result);
    
    // Check for kernel launch errors
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
    
    // Copy result back
    CUDA_CHECK(hipMemcpy(&h_result, d_result, sizeof(int), hipMemcpyDeviceToHost));
    
    // Free device memory
    CUDA_CHECK(hipFree(d_result));
    
    // Verify result
    if (h_result == 42) {
        std::cout << "✓ CUDA kernel execution successful!" << std::endl;
        std::cout << "✓ CUDA is properly configured and working!" << std::endl;
        return 0;
    } else {
        std::cerr << "✗ CUDA kernel execution failed!" << std::endl;
        return 1;
    }
}
